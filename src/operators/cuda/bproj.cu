
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define abs(x) (x > 0 ? x : -(x))
#define MAX(a,b) (a > b ? a : b)
#define MIN(a,b) (a < b ? a : b)
#define PI 3.1415926
#define GRIDDIM 32
#define BLOCKDIM 1024 //32*32


extern "C" void TOF_dist_backprojection(float *image_bp, const float *proj_value, const float *tof_value,
                            const float *x1l, const float *y1l, const float *x1r, const float *y1r,
                            const float *x2l, const float *y2l, const float *x2r, const float *y2r,
                            const float time_resolution, const float dx, const float dy,
                            const int nx, const int ny, const int event_num);

// define TOF-dist-bp
__device__ void TOF_dist_bp(float *image_bp, const float proj_value, const float tof_value,
                            const float x1l, const float y1l, const float x1r, const float y1r,
                            const float x2l, const float y2l, const float x2r, const float y2r,
                            const float time_resolution, const float dx, const float dy,
                            const int nx, const int ny)
{
    const float nx2 = nx/2;
    const float ny2 = ny/2;
    const float tof_sigma = time_resolution * 0.3 / 2.355 / 2;
    const float tof_sigma_2 = tof_sigma * tof_sigma;
    const float x1c = (x1l + x1r) / 2;
    const float y1c = (y1l + y1r) / 2;
    const float x2c = (x2l + x2r) / 2;
    const float y2c = (y2l + y2r) / 2;
    const float L = sqrtf((x1c - x2c) * (x1c - x2c) + (y1c - y2c) * (y1c - y2c));
    const float ratio1 = (1 - tof_value / L) / 2;

    const float xd = x1c - x2c;
    const float yd = y1c - y2c;
    float d2_tof, w_tof;
    if (abs(xd) > abs(yd))
    {
        for (int ix = 0; ix < nx; ix++)
        {
            float xc = (ix - nx2+0.5) * dx;
            float tof_bin = dx;
            
            if (tof_sigma > 0)
            {
                d2_tof = ((xc-x1c) / (x2c-x1c) - ratio1)*L;
                if (d2_tof <=3 * tof_sigma)
                {
                    w_tof = expf(-0.5 * d2_tof * d2_tof / tof_sigma_2) / sqrtf(2.0 * PI * tof_sigma_2) * tof_bin;
                }
                else
                {
                    w_tof = 0.0;
                }
            }
            else
            {
                w_tof = 1.0;
            }
            //d1l-d2r
            float kylr = (y1l-y2r)/(x1l-x2r);
            float ylr = kylr * (xc - x1l) + y1l + ny2 * dy;
            //d1r-d2l
            float kyrl = (y1r - y2l) / (x1r - x2l);
            float yrl = kyrl * (xc - x1r) + y1r + ny2 * dy;

            float yy1 = MIN(ylr,yrl); // 横坐标为xc时，detector边缘与x轴的交点中y较小值
            float yy2 = MAX(ylr,yrl);
            int cy1 = (int)floorf(yy1/dy);
            int cy2 = (int)floorf(yy2/dy);

            for (int iy=MAX(0, cy1); iy < MIN(ny, cy2+1); iy++)
            {
                float dist_w = (MIN((iy+1) * dy,yy2) - MAX(iy * dy,yy1)) / dy;
                atomicAdd(image_bp + (ix + iy * nx), proj_value * dist_w * w_tof);
            }

        }
        
    }
    else
    {
        for (int iy=0; iy < ny; iy++)
        {
            float yc = (iy - ny2 + 0.5) * dy;
            float tof_bin = dy;
            if (tof_sigma > 0)
            {
                d2_tof = ((yc-y1c) / (y2c-y1c) - ratio1) * L;
                if (d2_tof <=3 * tof_sigma)
                {
                    w_tof = expf(-0.5 * d2_tof * d2_tof / tof_sigma_2) / sqrtf(2.0 * PI * tof_sigma_2) * tof_bin;
                }
                else
                {
                    w_tof = 0.0;
                }
            }
            else
            {
                w_tof = 1.0;
            }
            //d1l-d2r:
            float kxlr = (x1l-x2r)/(y1l-y2r);
            float xlr = kxlr * (yc-y1l)+x1l+nx2 * dx;
            //d1r-d2l:
            float kxrl = (x1r-x2l)/(y1r-y2l);
            float xrl = kxrl * (yc-y1r)+x1r+nx2 * dx;
            
            float xx1 = MIN(xlr,xrl);
            float xx2 = MAX(xlr,xrl);
            float cx1 = (int)floorf(xx1/dx);
            float cx2 = (int)floorf(xx2/dx);

            
            for (int ix=MAX(0, cx1); ix < MIN(nx, cx2+1); ix++)
            {
                float dist_w = (MIN((ix+1) * dx,xx2) - MAX(ix * dx,xx1))/dx;
                atomicAdd(image_bp + (ix + iy * nx), proj_value * dist_w * w_tof);
            }

        }
    }

}

__global__ void TOF_dist_bp_kernel(float *image_bp, const float *proj_value, const float *tof_value,
                                   const float *x1l, const float *y1l, const float *x1r, const float *y1r,
                                   const float *x2l, const float *y2l, const float *x2r, const float *y2r,
                                   const float time_resolution, const float dx, const float dy,
                                   const int nx, const int ny, const int event_num)
{
    int step = blockDim.x * gridDim.x;
    for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < event_num; idx += step)
    {
        TOF_dist_bp(image_bp, proj_value[idx], tof_value[idx],
                    x1l[idx], y1l[idx], x1r[idx], y1r[idx],
                    x2l[idx], y2l[idx], x2r[idx], y2r[idx],
                    time_resolution, dx, dy,
                    nx, ny);
    }

}

void TOF_dist_backprojection(float *image_bp, const float *proj_value, const float *tof_value,
                            const float *x1l, const float *y1l, const float *x1r, const float *y1r,
                            const float *x2l, const float *y2l, const float *x2r, const float *y2r,
                            const float time_resolution, const float dx, const float dy,
                            const int nx, const int ny, const int event_num)
{
    float *image_bp_d, *proj_value_d, *tof_value_d, *x1l_d, *y1l_d, *x1r_d, *y1r_d, *x2l_d, *y2l_d, *x2r_d, *y2r_d;
    // 申请device内存
    hipMalloc(&image_bp_d, nx * ny * sizeof(float));
    hipMalloc(&proj_value_d, event_num * sizeof(float));
    hipMalloc(&tof_value_d, event_num * sizeof(float));
    hipMalloc(&x1l_d, event_num * sizeof(float));
    hipMalloc(&y1l_d, event_num * sizeof(float));
    hipMalloc(&x1r_d, event_num * sizeof(float));
    hipMalloc(&y1r_d, event_num * sizeof(float));
    hipMalloc(&x2l_d, event_num * sizeof(float));
    hipMalloc(&y2l_d, event_num * sizeof(float));
    hipMalloc(&x2r_d, event_num * sizeof(float));
    hipMalloc(&y2r_d, event_num * sizeof(float));
    // 将host上的数据拷贝到device上
    hipMemcpy(proj_value_d, proj_value, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tof_value_d, tof_value, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x1l_d, x1l, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y1l_d, y1l, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x1r_d, x1r, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y1r_d, x1r, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x2l_d, x2l, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y2l_d, y2l, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x2r_d, x2r, event_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y2r_d, x2r, event_num * sizeof(float), hipMemcpyHostToDevice);
    // 执行kernel
    TOF_dist_bp_kernel<<<GRIDDIM, BLOCKDIM>>>(image_bp_d, proj_value_d, tof_value_d, x1l_d, y1l_d, x1r_d, y1r_d,
                                            x2l_d, y2l_d, x2r_d, y2r_d, time_resolution, dx, dy, nx, ny, event_num);
    // 同步device，保证结果正常访问
    hipDeviceSynchronize();
    // 将device结果cpy到host
    hipMemcpy(image_bp, image_bp_d, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
    // 释放内存
    //cudaFree(image_bp_d);
    hipFree(proj_value_d);
    hipFree(tof_value_d);
    hipFree(x1l_d);
    hipFree(y1l_d);
    hipFree(x1r_d);
    hipFree(y1r_d);
    hipFree(x2l_d);
    hipFree(y2l_d);
    hipFree(x2r_d);
    hipFree(y2r_d);
}